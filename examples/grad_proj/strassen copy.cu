#include <iostream>
#include <thread>
#include <string>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "helper.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
using namespace std;

/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM kernel configurations (cutlass_tensorop_h16816gemm_128x128_32x4_nn_align8)
/////////////////////////////////////////////////////////////////////////////////////////////////

// A matrix configuration
using         ElementA    = float;//cutlass::half_t;                                // Element type for A matrix operand
using         LayoutA     = cutlass::layout::RowMajor;                      // Layout type for A matrix operand
constexpr int AlignmentA  = 128 / cutlass::sizeof_bits<ElementA>::value;    // Memory access granularity/alignment of A matrix in units of elements (up to 16 bytes)

// B matrix configuration
using         ElementB    = float;//cutlass::half_t;                                // Element type for B matrix operand
using         LayoutB     = cutlass::layout::RowMajor;                      // Layout type for B matrix operand
constexpr int AlignmentB  = 128 / cutlass::sizeof_bits<ElementB>::value;    // Memory access granularity/alignment of B matrix in units of elements (up to 16 bytes)

// C/D matrix configuration
using         ElementC    = float;//cutlass::half_t;                                // Element type for C and D matrix operands
using         LayoutC     = cutlass::layout::RowMajor;                      // Layout type for C and D matrix operands
constexpr int AlignmentC  = 128 / cutlass::sizeof_bits<ElementC>::value;    // Memory access granularity/alignment of C/D matrices in units of elements (up to 16 bytes)

// Multiply-accumulate blocking/pipelining details
using ElementAccumulator  = float;//cutlass::half_t;                          // Element type for internal accumulation
using ArchTag             = cutlass::arch::Sm80;                      // Tag indicating the minimum SM that supports the intended feature
using OperatorClass       = cutlass::arch::OpClassTensorOp;           // Operator class tag
using ThreadblockShape    = cutlass::gemm::GemmShape<128, 128, 16>;   // Threadblock-level tile size (concept: GemmShape)
using WarpShape           = cutlass::gemm::GemmShape<64, 64, 16>;     // Warp-level tile size (concept: GemmShape)
using InstructionShape    = cutlass::gemm::GemmShape<16, 8, 8>;       // Instruction-level tile size (concept: GemmShape)
constexpr int NumStages   = 4;                                        // Number of global->shared pipeline stages used in the GEMM mainloop

// Epilogue output operator
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementC,               // Element type for C and D matrix operands
    AlignmentC,             // Memory access granularity of C and D matrix in units of elements
    ElementAccumulator,     // Element type from internal accumaccumulation
    ElementAccumulator>;    // Data type used to compute linear combination

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;

// Classic data-parallel device GEMM implementation type
using DeviceGemmBasic = cutlass::gemm::device::GemmUniversal<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
    NumStages,
    AlignmentA,
    AlignmentB>;

// StreamK device GEMM implementation type
using DeviceGemmStreamK = cutlass::gemm::device::GemmUniversal<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueOp,
    cutlass::gemm::threadblock::ThreadblockSwizzleStreamK, // <-- Only difference
    NumStages,
    AlignmentA,
    AlignmentB>;


/////////////////////////////////////////////////////////////////////////////////////////////////
/// Testbed utility types
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Result structure
struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(true)
  {}

};


/// Command line options parsing
struct Options
{
  std::string               command_name;
  bool                      help;
  cutlass::gemm::GemmCoord  problem_size;
  float                     alpha;
  float                     beta;
  int                       split_k_factor;
  int                       avail_sms;
  bool                      reference_check;
  int                       iterations;

  cutlass::HostTensor<ElementA, LayoutA> tensor_a;
  cutlass::HostTensor<ElementB, LayoutB> tensor_b;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c;
  cutlass::HostTensor<ElementC, LayoutC> tensor_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_ref_d;

  Options(std::string command_name) :
    command_name(command_name),
    help(false),
    problem_size({2048, 2048, 2048}),
    alpha(1.0f),
    beta(0.0f),
    split_k_factor(1),
    avail_sms(-1),              // Number of device SMs to use is unlimited
    reference_check(true),
    iterations(10000)
  {}

  bool valid() const
  {
    return true;
  }

  void parse(int argc, char const **args)
  {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split", split_k_factor);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const
  {
    out
      << "Performs a GEMM computation.\n"
      << "\n"
      << "Options:\n"
      << "\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --split=<int>               Split-K factor to emulate\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << command_name << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    return 2.0 * double(problem_size.product()) / double(1.0e9) / runtime_s;
  }
};


/////////////////////////////////////////////////////////////////////////////////////////////////
/// GEMM evaluation
/////////////////////////////////////////////////////////////////////////////////////////////////

/// Populates a DeviceGemmBasic::Arguments structure from the given commandline options
typename DeviceGemmBasic::Arguments args_from_options(
    const DeviceGemmBasic &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d)
{
  return typename DeviceGemmBasic::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0));             // stride_d
}

/// Populates a DeviceGemmStreamK::Arguments structure from the given commandline options
typename DeviceGemmStreamK::Arguments args_from_options(
    const DeviceGemmStreamK &device_gemm,
    const Options &options,
    cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
    cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
    cutlass::HostTensor<ElementC, LayoutC> &tensor_d)
{
  return typename DeviceGemmStreamK::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0),              // stride_d
    options.avail_sms);                       // avail_sms
}

 Result run(Options &options)
 {
  // Instantiate CUTLASS kernel depending on templates
  DeviceGemmStreamK device_gemm;
  Result result;

  // Create a structure of gemm kernel arguments suitable for invoking an instance of DeviceGemmStreamK
  auto arguments = args_from_options(device_gemm, options, options.tensor_a, options.tensor_b, options.tensor_c, options.tensor_c);

  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = DeviceGemmStreamK::get_workspace_size(arguments);

  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // Check the problem size is supported or not
  CUTLASS_CHECK(device_gemm.can_implement(arguments));

  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));
  
  GpuTimer timer;
  timer.start();
  for (int iter = 0; iter < options.iterations; ++iter) {
    CUTLASS_CHECK(device_gemm());
  }
  timer.stop();
  // Compute average runtime and GFLOPs.
  float elapsed_ms = timer.elapsed_millis();
  result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
  result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);
  return result;
 }

float pos = 1.0f;
float neg = -1.0f;
int avail_sms = -1;

void strassen1(int m, int n, int k,
    float alpha,
     cutlass::TensorView<ElementA, LayoutA> &A, int ldA,
     cutlass::TensorView<ElementA, LayoutA> &B, int ldB,
    float beta,
     cutlass::TensorView<ElementA, LayoutA> &C, int ldC,
    float gamma,
     cutlass::TensorView<ElementA, LayoutA> &D
     ) {
      if (m <= 128 || n <= 128 || k <= 32) {
        DeviceGemmStreamK device_gemm;
        cutlass::TensorRef<ElementA, LayoutA> A_ref = A.ref();
        cutlass::TensorRef<ElementA, LayoutA> B_ref = B.ref();
        cutlass::TensorRef<ElementA, LayoutA> C_ref = C.ref();
        auto arguments = typename DeviceGemmStreamK::Arguments(
          cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
          {m, n, k},                     // problem_size
          1,                   // batch count / splitk slices
          {                                         // epilogue parameters
            ElementAccumulator(alpha),
            ElementAccumulator(beta)
          },
          A_ref.data(),                   // ptr_A
          B_ref.data(),                   // ptr_B
          C_ref.data(),                   // ptr_C
          C_ref.data(),                   // ptr_D
          m*k,      // batch_stride_A
          n*k,      // batch_stride_B
          m*n,      // batch_stride_C
          m*n,      // batch_stride_D
          A_ref.layout().stride(0),              // stride_a
          B_ref.layout().stride(0),              // stride_b
          C_ref.layout().stride(0),              // stride_c
          C_ref.layout().stride(0),              // stride_d
          avail_sms);                       // avail_sms
        size_t workspace_size = DeviceGemmStreamK::get_workspace_size(arguments);
        cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
        CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));
        CUTLASS_CHECK(device_gemm());
        return;
      }
      hipblasHandle_t handles[4];
      hipStream_t streams[4];
      for(int i = 0; i < 4; i++) {
        hipblasCreate(&handles[i]);
        hipStreamCreate(&streams[i]);
        hipblasSetStream(handles[i], streams[i]);
      }
      int subm = m / 2;
      int subn = n / 2;
      int subk = k / 2;
      cutlass::TensorView<ElementA, LayoutA> A0 = A.subview({subm, subk}, {0, 0});
      cutlass::TensorView<ElementA, LayoutA> A1 = A.subview({subm, subk}, {0, subk});
      cutlass::TensorView<ElementA, LayoutA> A2 = A.subview({subm, subk}, {subm, 0});
      cutlass::TensorView<ElementA, LayoutA> A3 = A.subview({subm, subk}, {subm, subk});
      cutlass::TensorView<ElementB, LayoutB> B0 = B.subview({subk, subn}, {0, 0});
      cutlass::TensorView<ElementB, LayoutB> B1 = B.subview({subk, subn}, {0, subn});
      cutlass::TensorView<ElementB, LayoutB> B2 = B.subview({subk, subn}, {subk, 0});
      cutlass::TensorView<ElementB, LayoutB> B3 = B.subview({subk, subn}, {subk, subn});
      cutlass::TensorView<ElementC, LayoutC> C0 = C.subview({subm, subn}, {0, 0});
      cutlass::TensorView<ElementC, LayoutC> C1 = C.subview({subm, subn}, {0, subn});
      cutlass::TensorView<ElementC, LayoutC> C2 = C.subview({subm, subn}, {subm, 0});
      cutlass::TensorView<ElementC, LayoutC> C3 = C.subview({subm, subn}, {subm, subn});
      cutlass::TensorRef<ElementA, LayoutA> A0_ref = A0.ref();
      cutlass::TensorRef<ElementA, LayoutA> A1_ref = A1.ref();
      cutlass::TensorRef<ElementA, LayoutA> A2_ref = A2.ref();
      cutlass::TensorRef<ElementA, LayoutA> A3_ref = A3.ref();
      cutlass::TensorRef<ElementB, LayoutB> B0_ref = B0.ref();
      cutlass::TensorRef<ElementB, LayoutB> B1_ref = B1.ref();
      cutlass::TensorRef<ElementB, LayoutB> B2_ref = B2.ref();
      cutlass::TensorRef<ElementB, LayoutB> B3_ref = B3.ref();
      cutlass::TensorRef<ElementC, LayoutC> C0_ref = C0.ref();
      cutlass::TensorRef<ElementC, LayoutC> C1_ref = C1.ref();
      cutlass::TensorRef<ElementC, LayoutC> C2_ref = C2.ref();
      cutlass::TensorRef<ElementC, LayoutC> C3_ref = C3.ref();

      hipblasSgeam(handles[0], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A0_ref.data(), subm, &pos, A1_ref.data(), subm, A1_ref.data(), subm);
      hipblasSgeam(handles[1], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A2_ref.data(), subm, &pos, A3_ref.data(), subm, A2_ref.data(), subm);
      hipblasSgeam(handles[2], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B1_ref.data(), subk, &neg, B3_ref.data(), subk, B1_ref.data(), subk);
      hipblasSgeam(handles[3], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B2_ref.data(), subk, &neg, B0_ref.data(), subk, B2_ref.data(), subk);
      hipStreamSynchronize (streams[0]);
      thread t1(strassen1, subm, subk, subn, 1.0f, std::ref(A1), subm, std::ref(B3), subk, 1.0f, std::ref(C0), subm, 1.0f, std::ref(C1));
      hipStreamSynchronize (streams[1]);
      thread t2(strassen1, subm, subk, subn, 1.0f, std::ref(A2), subm, std::ref(B0), subk, 1.0f, std::ref(C2), subm, -1.0f, std::ref(C3));
      hipStreamSynchronize (streams[2]);
      hipStreamSynchronize (streams[3]);
      t1.join();
      t2.join();
      hipblasSgeam(handles[0], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B0_ref.data(), subk, &pos, B3_ref.data(), subk, B0_ref.data(), subk);
      thread t3(strassen1, subm, subk, subn, 1, std::ref(A0), subm, std::ref(B1), subk, 1, std::ref(C1) , subm, 1, std::ref(C3));
      thread t4(strassen1, subm, subk, subn, 1, std::ref(A3), subm, std::ref(B2), subk, 1, std::ref(C0), subm, 1, std::ref(C2));
      t3.join();
      hipblasSgeam(handles[1], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A0_ref.data(), subm, &pos, A3_ref.data(), subm, A0_ref.data(), subm);
      hipStreamSynchronize (streams[0]);
      hipStreamSynchronize (streams[1]);
      t4.join();
      thread t5(strassen1, subm, subk, subn, 1, std::ref(A0), subm, std::ref(B0), subk, 1, std::ref(C0), subm, 1, std::ref(C3));
      hipblasSgeam(handles[0], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A2_ref.data(), subm, &neg, A0_ref.data(), subm, A2_ref.data(), subm);
      hipblasSgeam(handles[1], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A1_ref.data(), subm, &neg, A0_ref.data(), subm, A1_ref.data(), subm);
      hipblasSgeam(handles[2], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B1_ref.data(), subk, &pos, B0_ref.data(), subk, B1_ref.data(), subk);
      hipblasSgeam(handles[3], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B2_ref.data(), subk, &pos, B0_ref.data(), subk, B2_ref.data(), subk);
      hipStreamSynchronize (streams[0]);
      hipStreamSynchronize (streams[1]);
      hipStreamSynchronize (streams[2]);
      hipStreamSynchronize (streams[3]);
      t5.join();
      hipblasSgeam(handles[0], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A0_ref.data(), subm, &neg, A3_ref.data(), subm, A0_ref.data(), subm);
      hipblasSgeam(handles[1], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B0_ref.data(), subk, &neg, B3_ref.data(), subk, B0_ref.data(), subk); 
      thread t6(strassen1, subm, subk, subn, 1, std::ref(A2), subm, std::ref(B1), subk, 1, std::ref(C3), subm, 1, std::ref(C3));
      thread t7(strassen1, subm, subk, subn, 1, std::ref(A1), subm, std::ref(B2), subk, 1, std::ref(C0), subm, 1, std::ref(C0));
      hipStreamSynchronize (streams[0]);
      hipStreamSynchronize (streams[1]);
      t6.join();
      hipblasSgeam(handles[0], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A2_ref.data(), subm, &pos, A0_ref.data(), subm, A2_ref.data(), subm);
      hipblasSgeam(handles[1], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B1_ref.data(), subk, &neg, B0_ref.data(), subk, B1_ref.data(), subk); 
      t7.join();
      hipblasSgeam(handles[2], HIPBLAS_OP_T, HIPBLAS_OP_T, subm, subk, &pos, A1_ref.data(), subm, &pos, A3_ref.data(), subm, A1_ref.data(), subm);
      hipblasSgeam(handles[3], HIPBLAS_OP_T, HIPBLAS_OP_T, subk, subn, &pos, B2_ref.data(), subk, &neg, B3_ref.data(), subk, B2_ref.data(), subk); 
      hipStreamSynchronize (streams[0]);
      hipStreamSynchronize (streams[1]);
      hipStreamSynchronize (streams[2]);
      hipStreamSynchronize (streams[3]);
      for(int i = 0; i < 4; i++) {
        hipblasDestroy(handles[i]);
        hipStreamDestroy(streams[i]);
      }
      return;
}
 
 int test(Options options)
 {
   if (options.help) {
     options.print_usage(std::cout) << std::endl;
     return 0;
   }
 
   std::cout <<
     options.iterations << " timing iterations of " <<
     options.problem_size.m() << " x " <<
     options.problem_size.n() << " x " <<
     options.problem_size.k() << " matrix-matrix multiply" << std::endl;
 
   if (!options.valid()) {
     std::cerr << "Invalid problem." << std::endl;
     return -1;
   }
 
 
   //
   // Initialize GEMM datasets
   //
 
   // Initialize tensors using CUTLASS helper functions
   options.tensor_a.resize(options.problem_size.mk());       // <- Create matrix A with dimensions M x K
   options.tensor_b.resize(options.problem_size.kn());       // <- Create matrix B with dimensions K x N
   options.tensor_c.resize(options.problem_size.mn());       // <- Create matrix C with dimensions M x N

   // Fill matrix A on host with uniform-random data [-2, 2]
   cutlass::reference::host::TensorFillRandomUniform(
       options.tensor_a.host_view(),
       1,
       ElementA(2),
       ElementA(-2),
       0);
 
   // Fill matrix B on host with uniform-random data [-2, 2]
   cutlass::reference::host::TensorFillRandomUniform(
       options.tensor_b.host_view(),
       1,
       ElementB(2),
       ElementB(-2),
       0); 
 
   // Copy data from host to GPU
   options.tensor_a.sync_device();
   options.tensor_b.sync_device();
   GpuTimer timer;

   // CublasSgemm
   cutlass::reference::host::TensorFill(options.tensor_c.host_view());
   options.tensor_c.sync_device();
   hipblasHandle_t handle;
   hipblasCreate(&handle);
   timer.start();
   for (int i = 0; i < options.iterations; ++i) {
    hipblasStatus_t cublas_stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
      options.problem_size.m(), options.problem_size.n(), options.problem_size.k(),
      &options.alpha,
      options.tensor_a.device_data(), options.problem_size.m(),
      options.tensor_b.device_data(), options.problem_size.k(),
      &options.beta,
      options.tensor_c.device_data(), options.problem_size.m());
      if (cublas_stat != HIPBLAS_STATUS_SUCCESS) {
        cout << "cublassgemm failed with code: " << cublas_stat << endl;
        return -1;
      }
   }
   timer.stop();
   float elapsed_ms = timer.elapsed_millis();
   double cublas_time = double(elapsed_ms) / double(options.iterations);
   double culbas_gflops = options.gflops(cublas_time / 1000.0);
  printf("CUBLAS_GEMM:   [%6.1f]GFlop/s  (%6.4f)ms\n", culbas_gflops, cublas_time);
 
  //cutlass gemm
  cutlass::reference::host::TensorFill(options.tensor_c.host_view());
  options.tensor_c.sync_device();
  Result streamk_default  = run(options);
  printf("OPTIMIZED_GEMM:   [%6.1f]GFlop/s  (%6.4f)ms\n", streamk_default.gflops, streamk_default.avg_runtime_ms);

  //raw_strassen
  cutlass::reference::host::TensorFill(options.tensor_c.host_view());
  options.tensor_c.sync_device();


  //mixed_strassen 
  cutlass::reference::host::TensorFill(options.tensor_c.host_view());
  options.tensor_c.sync_device();

  timer
  timer.start();
  for (int iter = 0; iter < options.iterations; ++iter) {
    CUTLASS_CHECK(device_gemm());
  }
  timer.stop();
  // Compute average runtime and GFLOPs.
  float elapsed_ms = timer.elapsed_millis();
  result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
  result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);
  return result;

  hipblasDestroy(handle);
  return 0;
 }

int main(int argc, char const **argv) {
  // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
  if (!(__CUDACC_VER_MAJOR__ >= 11))
  {
  std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;

  // Returning zero so this test passes on older Toolkits. Its actions are no-op.
  return 0;
  }
  
  // Current device must must have compute capability at least 80
  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  if (!((props.major * 10 + props.minor) >= 80))
  {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
  
  // Parse commandline options
  #if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0

  Options options("Strassen matrix multiply");
  options.parse(argc, argv);
  test(options);
  return 0;

  #else

  std::cout << "Verification by comparison with cuBLAS is disabled, "
    "either because the CMake option CUTLASS_ENABLE_CUBLAS "
    "was explicitly set to OFF, or because CMake could not find cuBLAS. \n";

  #endif

}